#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

// CUDA Kernel for Vector Addition
__global__ void vectorAdd(float *A, float *B, float *C, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
    {
        C[i] = A[i] + B[i];
    }
}

int main()
{
    int N;
    cout << "Enter the number of elements in the vectors: ";
    cin >> N;

    // Size in bytes
    size_t size = N * sizeof(float);

    // Allocate memory on host
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // Initialize vectors on host
    for (int i = 0; i < N; i++)
    {
        h_A[i] = i * 1.0f;
        h_B[i] = i * 2.0f;
    }

    // Allocate memory on device
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // Copy vectors from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Launch the kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy result from device to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Print some of the results
    cout << "Result (first 10 elements):" << endl;
    for (int i = 0; i < min(N, 10); i++)
    {
        cout << h_A[i] << " + " << h_B[i] << " = " << h_C[i] << endl;
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
